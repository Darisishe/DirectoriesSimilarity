
#include <hip/hip_runtime.h>
#include <string>
#include <vector>

#define MAX_VALUE 256
#define BLOCKS 32
#define THREADS 128

__global__ void count(int *array, int *output, size_t n) {

  size_t b_id = blockIdx.x;
  size_t b_num = gridDim.x;
  size_t t_id = threadIdx.x;
  size_t t_num = blockDim.x;

  // initialize a shared memory array to store the count for each block.
  __shared__ int count[MAX_VALUE];

  // set intial values to zeros. Each thread sets its own share to zero.
  size_t t_size = (t_num > MAX_VALUE ? 1 : MAX_VALUE / t_num);
  size_t offset = t_id * t_size;
  for (size_t i = offset; i < offset + t_size && i < MAX_VALUE; ++i)
    count[i] = 0;

  // wait until all threads have completed the initialization process.
  __syncthreads();

  // accumulate the counts of each value. Each thread counts a certain portion
  // of the array.
  size_t b_size = (b_num > n ? 1 : n / b_num);
  size_t b_offset = b_id * b_size;

  t_size = (t_num > b_size ? 1 : b_size / t_num);

  offset = b_offset + t_id * t_size;
  for (size_t i = offset; i < offset + t_size && i < b_offset + b_size && i < n;
       ++i)
    atomicAdd(&count[array[i]], 1);

  // wait until all threads have completed the couting phase.
  __syncthreads();

  // copy the block count into global memory. Each thread copies its portion to
  // the global memory.
  t_size = (t_num > MAX_VALUE ? 1 : MAX_VALUE / t_num);
  size_t t_offset = t_id * t_size;
  offset = b_id * MAX_VALUE + t_offset;

  if (offset + t_size <= (b_id + 1) * MAX_VALUE)
    memcpy(&output[offset], &count[t_offset], sizeof(int) * t_size);
}

std::vector<size_t> count_symbols(const std::string &str) {
  size_t n = str.size();
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess || deviceCount == 0) {
    // If no GPU available, just run on CPU
    std::vector<size_t> counts(MAX_VALUE);
    for (auto c : str) {
      ++counts[static_cast<unsigned char>(c)];
    }

    return counts;
  
  } else {

    int *array = new int[n];
    for (size_t i = 0; i < n; ++i) {
      array[i] = static_cast<unsigned char>(str[i]);
    }

    int *d_array;
    int *d_output;
    hipMalloc((void **)&d_array, sizeof(int) * n);
    hipMalloc((void **)&d_output, sizeof(int) * BLOCKS * MAX_VALUE);

    hipMemcpy(d_array, array, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemset(d_output, 0, sizeof(int) * BLOCKS * MAX_VALUE);

    count<<<BLOCKS, THREADS>>>(d_array, d_output, n);

    int output[BLOCKS * MAX_VALUE];
    hipMemcpy(output, d_output, sizeof(int) * BLOCKS * MAX_VALUE,
               hipMemcpyDeviceToHost);

    // output contains chunks of counts
    std::vector<size_t> counts(MAX_VALUE);
    for (size_t i = 0; i < MAX_VALUE; ++i) {
      for (size_t j = 0; j < BLOCKS; ++j) {
        counts[i] += output[j * MAX_VALUE + i];
      }
    }

    hipFree(d_array);
    hipFree(d_output);

    delete[] array;

    return counts;
  }
}
